#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        std::cout << "Device Name: " << deviceProp.name << std::endl;
        std::cout << "Total Global Memory: " << deviceProp.totalGlobalMem << " bytes" << std::endl;
        // 打印其他设备属性...

        std::cout << std::endl;
    }

    return 0;
}
